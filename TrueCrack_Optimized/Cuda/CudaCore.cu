#include "hip/hip_runtime.h"
/*
 * Changes made 2012 Copyright (C)  Bastian Blankemeier
 * 									Henryk Jaskowiak
 *									Benjamin Meis
 *
 * Extending the original licensing of TrueCrack
 *
 *
 * TrueCrack Copyright (C) Luca Vaccaro
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "Tcdefs.h"
#include "Volumes.h"
#include <stdio.h>
#include <memory.h>
#include "Crypto.h"
#include "CudaCore.cuh"
#include "CudaPkcs5.cuh"
#include "CudaXts.cuh"



/* The max number of block grid; number of max parallel gpu blocks. */
int blockGridSizeMax;

/* The number of the current block grid; number of current parallel gpu blocks. */
int blockGridSizeCurrent;

/* Pointer of structures to pass to Cuda Kernel. */
unsigned char *dev_salt, *dev_blockPwd, *dev_header;
int *dev_blockPwd_init, *dev_blockPwd_length;
short int *dev_result;


int getMultiprocessorCount (int device_number){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,device_number); // 2nd parameter is the device number
	return prop.multiProcessorCount;
}

__global__ void cuda_Kernel ( unsigned char *salt, unsigned char *headerEncrypted, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, short int GPU_number, short int device_modulo, int offset) {

  	int numData=threadIdx.x + blockIdx.x * 512;
  	numData += offset; // Add offset

	// Array of unsigned char in the shared memory
	if (numData%device_modulo == GPU_number)
	{
	    __align__(8) unsigned char headerkey[192];
	    __align__(8) unsigned char headerDecrypted[512];
	
		// Calculate the hash header key
		for(int i = 0; i < 10; i++)
		{
			cuda_Pbkdf2 (salt, blockPwd, blockPwd_init, blockPwd_length, headerkey, numData, i);
		}
		
		// Decrypt the header and compare the key
	    
	    int value;
	    value=cuda_Xts (headerEncrypted, headerkey, headerDecrypted);
	
	    if (value==SUCCESS)
	        result[numData]=MATCH;
	    else
	        result[numData]=NOMATCH;
	}
}

void cuda_Core ( short int *result, short int GPU_number, short int device_modulo) {

	hipSetDevice(GPU_number);
	hipMalloc ( &dev_result, blockGridSizeCurrent * sizeof(short int)) ;
    hipMemcpy( dev_result, result, blockGridSizeCurrent * sizeof(short int) , hipMemcpyHostToDevice) ;

	int threadCount = 512;
	int gridSize = blockGridSizeCurrent / threadCount;
	cuda_Kernel<<<gridSize, threadCount>>>(dev_salt, dev_header, dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_result, GPU_number, device_modulo, 0);

	if (blockGridSizeCurrent % threadCount != 0)
	{
		int offset = gridSize * threadCount;
		gridSize = 1;
		threadCount = blockGridSizeCurrent % threadCount;
		hipDeviceSynchronize(); // Synchronize all Threads before starting next calculations
		cuda_Kernel<<<gridSize, threadCount>>>(dev_salt, dev_header, dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_result, GPU_number, device_modulo, offset);
	}

    //hipDeviceSynchronize();
    hipMemcpy(result, dev_result,blockGridSizeCurrent * sizeof(short int) , hipMemcpyDeviceToHost) ;
}

/*The reference manual lists all the various functions used to copy memory between 
linear memory allocated with hipMalloc(), linear memory allocated with 
hipMallocPitch() or hipMalloc3D(), CUDA arrays, and memory 
allocated for variables declared in global or constant memory space.
The following code sample illustrates various ways of accessing global variables via 
the runtime API:
__constant__ float constData[256];
float data[256];
hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));
__device__ float devData;
float value = 3.14f;
hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
__device__ float* devPointer;
float* ptr;
hipMalloc(&ptr, 256 * sizeof(float));
hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));
hipGetSymbolAddress() is used to retrieve the address pointing to the 
memory allocated for a variable declared in global memory space. The size of the 
allocated memory is obtained through hipGetSymbolSize().*/

void cuda_Init (int block_maxsize, unsigned char *salt, unsigned char *header, int device_number) {
    blockGridSizeMax=block_maxsize;
    hipSetDevice(device_number);
    hipMalloc ( (void **)&dev_blockPwd, blockGridSizeMax * PASSWORD_MAXSIZE * sizeof(unsigned char)) ;
    hipMalloc ( (void **)&dev_blockPwd_init, blockGridSizeMax * sizeof(int)) ;
    hipMalloc ( (void **)&dev_blockPwd_length, blockGridSizeMax * sizeof(int)) ;
    hipMalloc ( (void **)&dev_salt, SALT_LENGTH * sizeof(unsigned char)) ;
    hipMalloc ( (void **)&dev_header, TC_VOLUME_HEADER_EFFECTIVE_SIZE * sizeof(unsigned char)) ;
    hipMalloc ( (void **)&dev_result, blockGridSizeMax * sizeof(short int)) ;

    hipMemcpy(dev_salt, salt, SALT_LENGTH * sizeof(unsigned char) , hipMemcpyHostToDevice) ;
    hipMemcpy(dev_header, header, TC_VOLUME_HEADER_EFFECTIVE_SIZE * sizeof(unsigned char) , hipMemcpyHostToDevice) ;

}

void cuda_Set (	int block_currentsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int device_number) {

	hipSetDevice(device_number);
	blockGridSizeCurrent=block_currentsize;
	int lengthpwd=0,i;
	for (i=0;i<blockGridSizeCurrent;i++) {
		lengthpwd+=blockPwd_length[i];
		result[i]=NODEFINED;
	}

	hipMemcpy(dev_blockPwd, blockPwd, lengthpwd * sizeof(unsigned char) , hipMemcpyHostToDevice) ;
	hipMemcpy(dev_blockPwd_init, blockPwd_init, blockGridSizeCurrent * sizeof(int) , hipMemcpyHostToDevice) ;
	hipMemcpy(dev_blockPwd_length, blockPwd_length, blockGridSizeCurrent * sizeof(int) , hipMemcpyHostToDevice) ;
	hipMemcpy(dev_result, result, blockGridSizeCurrent * sizeof(short int) , hipMemcpyHostToDevice) ;
}


void cuda_Free(void) {
    hipFree(dev_salt);
    hipFree(dev_blockPwd);
    hipFree(dev_blockPwd_init);
    hipFree(dev_blockPwd_length);
    hipFree(dev_result);
    hipFree(dev_header);
}
